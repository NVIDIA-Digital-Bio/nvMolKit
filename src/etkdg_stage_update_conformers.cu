// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
// SPDX-License-Identifier: Apache-2.0
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <GraphMol/ROMol.h>

#include "etkdg_stage_update_conformers.h"

namespace nvMolKit {
namespace detail {

ETKDGUpdateConformersStage::ETKDGUpdateConformersStage(const std::vector<RDKit::ROMol*>& mols,
                                                       const std::vector<EmbedArgs>&     eargs,
                                                       hipStream_t                      stream,
                                                       std::mutex*                       conformer_mutex,
                                                       int                               maxConformersPerMol)
    : mols_(mols),
      eargs_(eargs),
      stream_(stream),
      conformer_mutex_(conformer_mutex),
      maxConformersPerMol_(maxConformersPerMol) {}

void ETKDGUpdateConformersStage::execute(ETKDGContext& ctx) {
  // Copy positions from device to host
  std::vector<double> hostPositions;
  hostPositions.resize(ctx.systemDevice.positions.size());
  ctx.systemDevice.positions.copyToHost(hostPositions);

  // Copy active this stage from device to host
  std::vector<uint8_t> hostActiveThisStage;
  hostActiveThisStage.resize(ctx.activeThisStage.size());
  ctx.activeThisStage.copyToHost(hostActiveThisStage);
  hipStreamSynchronize(stream_);

  // Update each molecule's conformer with the new positions
  for (size_t i = 0; i < mols_.size(); ++i) {
    // Skip if not active this stage
    if (hostActiveThisStage[i] != 1) {
      continue;
    }

    const auto& mol         = mols_[i];
    const int   dim         = eargs_[i].dim;
    const int   startPosIdx = ctx.systemHost.atomStarts[i] * dim;
    const int   nAtoms      = mol->getNumAtoms();

    auto newConf = std::make_unique<RDKit::Conformer>(mol->getNumAtoms());

    for (int j = 0; j < nAtoms; ++j) {
      const int       posIdx = startPosIdx + j * dim;
      RDGeom::Point3D pos(hostPositions[posIdx], hostPositions[posIdx + 1], hostPositions[posIdx + 2]);
      newConf->setAtomPos(j, pos);
    }

    // Thread-safe conformer addition with count checking
    if (conformer_mutex_) {
      std::lock_guard<std::mutex> lock(*conformer_mutex_);
      // Check if molecule already has enough conformers
      if (maxConformersPerMol_ <= 0 || mol->getNumConformers() < static_cast<unsigned int>(maxConformersPerMol_)) {
        mol->addConformer(newConf.release(), /*assignId=*/true);  // true to take ownership
      }
    } else {
      // Without mutex, assume single-threaded and add conformer
      mol->addConformer(newConf.release(), /*assignId=*/true);
    }
    // If conformer wasn't added, it's still a unique_ptr, and will destruct out of scope.
  }
}

}  // namespace detail
}  // namespace nvMolKit
