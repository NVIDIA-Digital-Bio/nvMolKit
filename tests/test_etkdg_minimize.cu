// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
// SPDX-License-Identifier: Apache-2.0
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <DistGeom/BoundsMatrix.h>
#include <DistGeom/ChiralSet.h>
#include <ForceField/ForceField.h>
#include <gmock/gmock.h>
#include <GraphMol/DistGeomHelpers/Embedder.h>
#include <GraphMol/FileParsers/FileParsers.h>
#include <gtest/gtest.h>

#include <filesystem>

#include "dist_geom.h"
#include "embedder_utils.h"
#include "etkdg_impl.h"
#include "etkdg_stage_coordgen.h"
#include "etkdg_stage_firstminimization.h"
#include "etkdg_stage_fourthdimminimization.h"
#include "test_utils.h"

using namespace ::nvMolKit::detail;

using ETKDGStageTestParams = std::tuple<ETKDGOption, int>;

namespace {

// Helper function for common initialization logic
void initTestComponentsCommon(const std::vector<const RDKit::ROMol*>&           mols,
                              const std::vector<std::unique_ptr<RDKit::RWMol>>& molsPtrs,
                              ETKDGContext&                                     context,
                              std::vector<nvMolKit::detail::EmbedArgs>&         eargsVec,
                              RDKit::DGeomHelpers::EmbedParameters&             embedParam) {
  // Initialize context
  context.nTotalSystems         = mols.size();
  context.systemHost.atomStarts = {0};
  context.systemHost.positions.clear();

  // Process each molecule
  for (size_t i = 0; i < mols.size(); ++i) {
    nvMolKit::detail::EmbedArgs eargs;
    eargs.dim = 4;
    std::vector<std::unique_ptr<RDGeom::Point>> positions;
    std::unique_ptr<ForceFields::ForceField>    field;

    // Setup force field and get parameters
    nvMolKit::DGeomHelpers::setupRDKitFFWithPos(molsPtrs[i].get(),
                                                embedParam,
                                                field,
                                                eargs,
                                                positions,
                                                -1,  // Use default conformer
                                                nvMolKit::DGeomHelpers::Dimensionality::DIM_4D);

    // Add molecule to context with positions
    nvMolKit::DistGeom::addMoleculeToContextWithPositions(eargs.posVec,
                                                          eargs.dim,
                                                          context.systemHost.atomStarts,
                                                          context.systemHost.positions);

    // Store embed args for later use
    eargsVec.push_back(std::move(eargs));
  }

  // Send context to device
  nvMolKit::DistGeom::sendContextToDevice(context.systemHost.positions,
                                          context.systemDevice.positions,
                                          context.systemHost.atomStarts,
                                          context.systemDevice.atomStarts);
}

// Helper function to calculate initial energies for one or more molecules
std::vector<double> calculateInitialEnergies(const std::vector<std::unique_ptr<RDKit::RWMol>>& mols) {
  std::vector<double> initialEnergies;
  initialEnergies.reserve(mols.size());
  auto params = RDKit::DGeomHelpers::ETKDGv3;
  for (size_t i = 0; i < mols.size(); ++i) {
    nvMolKit::detail::EmbedArgs                 eargs;
    std::vector<std::unique_ptr<RDGeom::Point>> positions;
    std::unique_ptr<ForceFields::ForceField>    field;
    nvMolKit::DGeomHelpers::setupRDKitFFWithPos(mols[i].get(), params, field, eargs, positions);
    const double initialEnergy = field->calcEnergy();
    EXPECT_GE(initialEnergy, 0.0) << "Initial energy should be non-negative for molecule " << i;
    initialEnergies.push_back(initialEnergy);
  }

  return initialEnergies;
}

// Helper function to check final energies against initial energies and failure counts
void checkFinalEnergies(const std::vector<double>&                        finalEnergies,
                        const std::vector<double>&                        initialEnergies,
                        const std::vector<std::unique_ptr<RDKit::RWMol>>& mols,
                        const std::vector<std::vector<int16_t>>&          failureCounts,
                        const std::string&                                context = "") {
  for (size_t i = 0; i < mols.size(); ++i) {
    // Skip if initial energy is 0 or molecule failed in any stage
    bool shouldSkip = (initialEnergies[i] == 0.0);
    for (const auto& stageFailures : failureCounts) {
      if (stageFailures[i] > 0) {
        shouldSkip = true;
        break;
      }
    }
    if (shouldSkip) {
      continue;
    }

    EXPECT_LT(finalEnergies[i], initialEnergies[i])
      << context << "Molecule " << i << ": Final energy (" << finalEnergies[i]
      << ") should be less than initial energy (" << initialEnergies[i] << ")";

    const int    numAtoms      = mols[i]->getNumAtoms();
    const double energyPerAtom = finalEnergies[i] / numAtoms;
    EXPECT_LT(energyPerAtom, nvMolKit::detail::MAX_MINIMIZED_E_PER_ATOM)
      << context << "Molecule " << i << ": Energy per atom (" << energyPerAtom << ") should be below threshold ("
      << nvMolKit::detail::MAX_MINIMIZED_E_PER_ATOM << ")";
  }
}

}  // anonymous namespace

// Test fixture for single molecule tests
class ETKDGMinimizeSingleMolTestFixture : public ::testing::TestWithParam<ETKDGOption> {
 public:
  ETKDGMinimizeSingleMolTestFixture() { testDataFolderPath_ = getTestDataFolderPath(); }

  void SetUp() override {
    // Load molecule
    const std::string mol2FilePath = testDataFolderPath_ + "/rdkit_smallmol_1.mol2";
    ASSERT_TRUE(std::filesystem::exists(mol2FilePath)) << "Could not find " << mol2FilePath;
    molPtr_ = std::unique_ptr<RDKit::RWMol>(RDKit::MolFileToMol(mol2FilePath, false));
    ASSERT_NE(molPtr_, nullptr);
    molPtr_->clearConformers();
    RDKit::MolOps::sanitizeMol(*molPtr_);

    // Initialize mols_ vector with the single molecule
    mols_.push_back(molPtr_.get());

    // Initialize molsPtrs_ vector for the common function
    molsPtrs_.push_back(std::move(molPtr_));

    // Initialize common test components
    embedParam_                 = getETKDGOption(GetParam());
    embedParam_.useRandomCoords = true;
    initTestComponents();
  }

  void initTestComponents() { initTestComponentsCommon(mols_, molsPtrs_, context_, eargs_, embedParam_); }

 protected:
  std::string                                testDataFolderPath_;
  std::unique_ptr<RDKit::RWMol>              molPtr_;
  std::vector<std::unique_ptr<RDKit::RWMol>> molsPtrs_;
  std::vector<const RDKit::ROMol*>           mols_;
  ETKDGContext                               context_;
  std::vector<nvMolKit::detail::EmbedArgs>   eargs_;
  RDKit::DGeomHelpers::EmbedParameters       embedParam_;
};

// BFGS Stage Tests
TEST_P(ETKDGMinimizeSingleMolTestFixture, FirstMinimizeStageBFGSTest) {
  // Calculate initial energy
  const std::vector<double> initialEnergies = calculateInitialEnergies(molsPtrs_);

  // Create FirstMinimizeStage
  std::vector<std::unique_ptr<ETKDGStage>> stages;
  auto  stage    = std::make_unique<nvMolKit::detail::FirstMinimizeStage>(mols_, eargs_, embedParam_, context_);
  auto* stagePtr = stage.get();  // Store pointer before moving
  stages.push_back(std::move(stage));

  // Create and run driver
  ETKDGDriver driver(std::make_unique<ETKDGContext>(std::move(context_)), std::move(stages));
  driver.run(5);

  // Get final energy from the stage
  std::vector<double> finalEnergies(stagePtr->molSystemDevice.energyOuts.size());
  stagePtr->molSystemDevice.energyOuts.copyToHost(finalEnergies);

  // Check other results first
  EXPECT_EQ(driver.numConfsFinished(), 1);
  EXPECT_EQ(driver.iterationsComplete(), 1);

  auto failureCounts = driver.getFailures();
  EXPECT_EQ(failureCounts.size(), 1);                      // One stage
  EXPECT_THAT(failureCounts[0], testing::ElementsAre(0));  // FirstMinimizeStage

  auto completed = driver.completedConformers();
  EXPECT_THAT(completed, testing::ElementsAre(1));

  // Check energy reduction and threshold
  checkFinalEnergies(finalEnergies, initialEnergies, molsPtrs_, failureCounts);
}

TEST_P(ETKDGMinimizeSingleMolTestFixture, FourthDimMinimizeStageBFGSTest) {
  // Create FourthDimMinimizeStage
  std::vector<std::unique_ptr<ETKDGStage>> stages;
  stages.push_back(std::make_unique<nvMolKit::detail::FourthDimMinimizeStage>(mols_, eargs_, embedParam_, context_));

  // Create and run driver
  ETKDGDriver driver(std::make_unique<ETKDGContext>(std::move(context_)), std::move(stages));
  driver.run(5);

  // Check results
  EXPECT_EQ(driver.numConfsFinished(), 1);
  EXPECT_EQ(driver.iterationsComplete(), 1);

  auto failureCounts = driver.getFailures();
  EXPECT_EQ(failureCounts.size(), 1);                      // One stage
  EXPECT_THAT(failureCounts[0], testing::ElementsAre(0));  // FourthDimMinimizeStage

  auto completed = driver.completedConformers();
  EXPECT_THAT(completed, testing::ElementsAre(1));
}

TEST_P(ETKDGMinimizeSingleMolTestFixture, FullMinimizationPipelineBFGSTest) {
  // Calculate initial energy
  const std::vector<double> initialEnergies = calculateInitialEnergies(molsPtrs_);

  // Create stages
  std::vector<std::unique_ptr<ETKDGStage>> stages;
  auto  firstStage    = std::make_unique<nvMolKit::detail::FirstMinimizeStage>(mols_, eargs_, embedParam_, context_);
  auto* firstStagePtr = firstStage.get();  // Store pointer before moving
  stages.push_back(std::move(firstStage));
  stages.push_back(std::make_unique<nvMolKit::detail::FourthDimMinimizeStage>(mols_, eargs_, embedParam_, context_));

  // Create and run driver
  ETKDGDriver driver(std::make_unique<ETKDGContext>(std::move(context_)), std::move(stages));
  driver.run(5);

  // Get final energy from the first stage
  std::vector<double> finalEnergies(firstStagePtr->molSystemDevice.energyOuts.size());
  firstStagePtr->molSystemDevice.energyOuts.copyToHost(finalEnergies);

  // Check other results first
  EXPECT_EQ(driver.numConfsFinished(), 1);
  EXPECT_EQ(driver.iterationsComplete(), 1);

  auto failureCounts = driver.getFailures();
  EXPECT_EQ(failureCounts.size(), 2);                      // Two stages
  EXPECT_THAT(failureCounts[0], testing::ElementsAre(0));  // FirstMinimizeStage
  EXPECT_THAT(failureCounts[1], testing::ElementsAre(0));  // FourthDimMinimizeStage

  auto completed = driver.completedConformers();
  EXPECT_THAT(completed, testing::ElementsAre(1));

  // Check energy reduction and threshold
  checkFinalEnergies(finalEnergies, initialEnergies, molsPtrs_, failureCounts);
}

TEST_P(ETKDGMinimizeSingleMolTestFixture, FirstPartETKDGPipelineBFGSTest) {
  constexpr int16_t         maxFailedIterations = 2;
  // Calculate initial energy
  const std::vector<double> initialEnergies     = calculateInitialEnergies(molsPtrs_);

  // Zero out positions on device since we are using coordgen stage for generating initial coordinates
  context_.systemDevice.positions.zero();

  // Create stages in order: coordgen -> first minimize BFGS -> fourthdim BFGS
  std::vector<std::unique_ptr<ETKDGStage>> stages;
  stages.push_back(std::make_unique<nvMolKit::detail::ETKDGCoordGenRDKitStage>(embedParam_, mols_, eargs_));
  auto  firstStage    = std::make_unique<nvMolKit::detail::FirstMinimizeStage>(mols_, eargs_, embedParam_, context_);
  auto* firstStagePtr = firstStage.get();  // Store pointer before moving
  stages.push_back(std::move(firstStage));
  stages.push_back(std::make_unique<nvMolKit::detail::FourthDimMinimizeStage>(mols_, eargs_, embedParam_, context_));

  // Create and run driver
  ETKDGDriver driver(std::make_unique<ETKDGContext>(std::move(context_)), std::move(stages));
  driver.run(3);

  // Get final energy from the first stage
  std::vector<double> finalEnergies(firstStagePtr->molSystemDevice.energyOuts.size());
  firstStagePtr->molSystemDevice.energyOuts.copyToHost(finalEnergies);

  // Check other results first
  EXPECT_EQ(driver.numConfsFinished(), 1);
  EXPECT_LE(driver.iterationsComplete(), 2);  // Allow for 1 failure.

  auto failureCounts = driver.getFailures();
  EXPECT_EQ(failureCounts.size(), 3);                                              // Three stages
  EXPECT_THAT(failureCounts[0], testing::Each(0));                                 // CoordGenStage
  EXPECT_THAT(failureCounts[1], testing::Each(testing::Le(maxFailedIterations)));  // FirstMinimizeStage
  EXPECT_THAT(failureCounts[2], testing::Each(testing::Le(maxFailedIterations)));  // FourthDimMinimizeStage

  auto completed = driver.completedConformers();
  EXPECT_THAT(completed, testing::ElementsAre(1));

  // Check energy reduction and threshold
  checkFinalEnergies(finalEnergies, initialEnergies, molsPtrs_, failureCounts);
}

// Test fixture for multiple diverse molecules tests
class ETKDGMinimizeMultiMolDiverseTestFixture : public ::testing::TestWithParam<ETKDGOption> {
 public:
  ETKDGMinimizeMultiMolDiverseTestFixture() { testDataFolderPath_ = getTestDataFolderPath(); }

  void SetUp() override {
    // Load multiple different molecules from MMFF94_dative.sdf
    std::vector<std::unique_ptr<RDKit::ROMol>> tempMols;
    getMols(testDataFolderPath_ + "/MMFF94_dative.sdf", tempMols, /*count=*/5);
    ASSERT_EQ(tempMols.size(), 5) << "Expected to load 5 molecules";

    // Convert to RWMol and prepare molecules
    for (auto& tempMol : tempMols) {
      molsPtrs_.push_back(std::make_unique<RDKit::RWMol>(*tempMol));
    }

    // Clear conformers and sanitize all molecules and prepare mols_ vector with pointers
    for (auto& molPtr : molsPtrs_) {
      molPtr->clearConformers();
      RDKit::MolOps::sanitizeMol(*molPtr);
      mols_.push_back(molPtr.get());
    }
    ASSERT_EQ(mols_.size(), 5) << "Expected 5 molecules";

    // Initialize common test components
    embedParam_                 = getETKDGOption(GetParam());
    embedParam_.useRandomCoords = true;
    initTestComponents();
  }

  void initTestComponents() { initTestComponentsCommon(mols_, molsPtrs_, context_, eargs_, embedParam_); }

 protected:
  std::string                                testDataFolderPath_;
  std::vector<std::unique_ptr<RDKit::RWMol>> molsPtrs_;
  std::vector<const RDKit::ROMol*>           mols_;
  ETKDGContext                               context_;
  std::vector<nvMolKit::detail::EmbedArgs>   eargs_;
  RDKit::DGeomHelpers::EmbedParameters       embedParam_;
};

// BFGS Stage Tests for diverse molecules
TEST_P(ETKDGMinimizeMultiMolDiverseTestFixture, FirstMinimizeStageBFGSTest) {
  constexpr int16_t maxFailedIterations = 2;

  // Calculate initial energies for all molecules
  const std::vector<double> initialEnergies = calculateInitialEnergies(molsPtrs_);

  // Create FirstMinimizeStage
  std::vector<std::unique_ptr<ETKDGStage>> stages;
  auto  stage    = std::make_unique<nvMolKit::detail::FirstMinimizeStage>(mols_, eargs_, embedParam_, context_);
  auto* stagePtr = stage.get();  // Store pointer before moving
  stages.push_back(std::move(stage));

  // Create and run driver
  ETKDGDriver driver(std::make_unique<ETKDGContext>(std::move(context_)), std::move(stages));
  driver.run(2);

  // Get final energies from the stage
  std::vector<double> finalEnergies(stagePtr->molSystemDevice.energyOuts.size());
  stagePtr->molSystemDevice.energyOuts.copyToHost(finalEnergies);

  // Get failure counts
  auto failureCounts = driver.getFailures();
  EXPECT_EQ(failureCounts.size(), 1);                                              // One stage
  EXPECT_THAT(failureCounts[0], testing::Each(testing::Le(maxFailedIterations)));  // FirstMinimizeStage

  // Check other results
  EXPECT_GE(driver.numConfsFinished(), 3);
  EXPECT_LE(driver.iterationsComplete(), 2);

  auto completed = driver.completedConformers();
  EXPECT_THAT(completed, testing::Each(testing::AnyOf(0, 1)));

  // Check energy reduction and threshold for each molecule
  checkFinalEnergies(finalEnergies, initialEnergies, molsPtrs_, failureCounts);
}

TEST_P(ETKDGMinimizeMultiMolDiverseTestFixture, FourthDimMinimizeStageBFGSTest) {
  // Create FourthDimMinimizeStage
  std::vector<std::unique_ptr<ETKDGStage>> stages;
  stages.push_back(std::make_unique<nvMolKit::detail::FourthDimMinimizeStage>(mols_, eargs_, embedParam_, context_));

  // Create and run driver
  ETKDGDriver driver(std::make_unique<ETKDGContext>(std::move(context_)), std::move(stages));
  driver.run(2);

  // Check results
  EXPECT_EQ(driver.numConfsFinished(), 5);
  EXPECT_EQ(driver.iterationsComplete(), 1);

  auto failureCounts = driver.getFailures();
  EXPECT_EQ(failureCounts.size(), 1);               // One stage
  EXPECT_THAT(failureCounts[0], testing::Each(0));  // FourthDimMinimizeStage

  auto completed = driver.completedConformers();
  EXPECT_THAT(completed, testing::Each(1));
}

TEST_P(ETKDGMinimizeMultiMolDiverseTestFixture, FullMinimizationPipelineBFGSTest) {
  constexpr int16_t maxFailedIterations = 2;

  // Calculate initial energies for all molecules
  const std::vector<double> initialEnergies = calculateInitialEnergies(molsPtrs_);

  // Create stages
  std::vector<std::unique_ptr<ETKDGStage>> stages;
  auto  firstStage    = std::make_unique<nvMolKit::detail::FirstMinimizeStage>(mols_, eargs_, embedParam_, context_);
  auto* firstStagePtr = firstStage.get();  // Store pointer before moving
  stages.push_back(std::move(firstStage));
  auto secondStage = std::make_unique<nvMolKit::detail::FourthDimMinimizeStage>(mols_, eargs_, embedParam_, context_);
  auto secondStagePtr = secondStage.get();
  stages.push_back(std::move(secondStage));

  // Create and run driver
  ETKDGDriver driver(std::make_unique<ETKDGContext>(std::move(context_)), std::move(stages));
  driver.run(2);

  // Get final energies from the first stage
  std::vector<double> finalEnergies(secondStagePtr->molSystemDevice.energyOuts.size());
  secondStagePtr->molSystemDevice.energyOuts.copyToHost(finalEnergies);

  // Get failure counts
  auto failureCounts = driver.getFailures();
  EXPECT_EQ(failureCounts.size(), 2);                                              // Two stages
  EXPECT_THAT(failureCounts[0], testing::Each(testing::Le(maxFailedIterations)));  // FirstMinimizeStage
  EXPECT_THAT(failureCounts[1], testing::Each(testing::Le(maxFailedIterations)));  // FourthDimMinimizeStage

  // Check other results
  EXPECT_GE(driver.numConfsFinished(), 3);
  EXPECT_LE(driver.iterationsComplete(), 2);

  auto completed = driver.completedConformers();
  EXPECT_THAT(completed, testing::Each(testing::AnyOf(0, 1)));

  // Check energy reduction and threshold for each molecule
  checkFinalEnergies(finalEnergies, initialEnergies, molsPtrs_, failureCounts);
}

TEST_P(ETKDGMinimizeMultiMolDiverseTestFixture, FirstPartETKDGPipelineBFGSTest) {
  constexpr int16_t         maxFailedIterations = 2;
  // Calculate initial energies for all molecules
  const std::vector<double> initialEnergies     = calculateInitialEnergies(molsPtrs_);

  // Zero out positions on device since we are using coordgen stage for generating initial coordinates
  context_.systemDevice.positions.zero();

  // Create stages in order: coordgen -> first minimize BFGS -> fourthdim BFGS
  std::vector<std::unique_ptr<ETKDGStage>> stages;
  stages.push_back(std::make_unique<nvMolKit::detail::ETKDGCoordGenRDKitStage>(embedParam_, mols_, eargs_));
  auto firstStage = std::make_unique<nvMolKit::detail::FirstMinimizeStage>(mols_, eargs_, embedParam_, context_);
  stages.push_back(std::move(firstStage));
  auto  secondStage = std::make_unique<nvMolKit::detail::FourthDimMinimizeStage>(mols_, eargs_, embedParam_, context_);
  auto* secondStagePtr = secondStage.get();  // Store pointer before moving
  stages.push_back(std::move(secondStage));
  // Create and run driver
  ETKDGDriver driver(std::make_unique<ETKDGContext>(std::move(context_)), std::move(stages));
  driver.run(3);
  auto failureCounts = driver.getFailures();

  // Get final energies from the first stage
  std::vector<double> finalEnergies(secondStagePtr->molSystemDevice.energyOuts.size());
  secondStagePtr->molSystemDevice.energyOuts.copyToHost(finalEnergies);
  hipDeviceSynchronize();

  // Get failure counts
  EXPECT_EQ(failureCounts.size(), 3);                                              // Three stages
  EXPECT_THAT(failureCounts[0], testing::Each(0));                                 // CoordGenStage
  EXPECT_THAT(failureCounts[1], testing::Each(testing::Le(maxFailedIterations)));  // FirstMinimizeStage
  EXPECT_THAT(failureCounts[2], testing::Each(testing::Le(maxFailedIterations)));  // FourthDimMinimizeStage

  // Check other results
  EXPECT_GE(driver.numConfsFinished(), 3);
  EXPECT_LE(driver.iterationsComplete(), 3);

  // Check energy reduction and threshold for each molecule
  checkFinalEnergies(finalEnergies, initialEnergies, molsPtrs_, failureCounts);
}

TEST_P(ETKDGMinimizeMultiMolDiverseTestFixture, FirstMinimizeStageBFGSWithInactiveMolecules) {
  // Create FirstMinimizeStage
  auto stage = std::make_unique<nvMolKit::detail::FirstMinimizeStage>(mols_, eargs_, embedParam_, context_);

  // Set some molecules as inactive (let's say molecules 1 and 3)
  std::vector<uint8_t> activeRef(context_.nTotalSystems, 1);
  activeRef[1] = 0;  // Mark second molecule as inactive
  activeRef[3] = 0;  // Mark fourth molecule as inactive
  context_.activeThisStage.resize(context_.nTotalSystems);
  context_.activeThisStage.copyFromHost(activeRef);
  context_.failedThisStage.resize(context_.nTotalSystems);

  // Execute the stage
  stage->execute(context_);

  // Copy energy outputs from device to host
  std::vector<double> energyOuts(stage->molSystemDevice.energyOuts.size());
  stage->molSystemDevice.energyOuts.copyToHost(energyOuts);

  // Check that inactive molecules have zero energy
  for (int i = 0; i < context_.nTotalSystems; ++i) {
    if (activeRef[i] == 0) {
      // Empirical observation: inactive molecules typically have energy > 50
      // since they haven't been minimized by BFGS
      EXPECT_GT(energyOuts[i], 50.0) << "Inactive molecule " << i
                                     << " should have high energy (>50) since it wasn't minimized";
    } else {
      // Empirical observation: active molecules typically have energy < 0.1
      // after successful BFGS minimization
      EXPECT_LT(energyOuts[i], 0.1) << "Active molecule " << i
                                    << " should have low energy (<0.1) after successful minimization";
    }
  }
}

// Instantiate test suites for both fixtures
INSTANTIATE_TEST_SUITE_P(
  ETKDGOptions,
  ETKDGMinimizeSingleMolTestFixture,
  ::testing::Values(ETKDGOption::ETKDGv3, ETKDGOption::ETKDGv2, ETKDGOption::ETKDG, ETKDGOption::KDG),
  [](const ::testing::TestParamInfo<ETKDGOption>& info) { return getETKDGOptionName(info.param); });

// TODO: Currently only testing ETKDGv3 due to non-deterministic failures when testing multiple options.
// When multiple ETKDGOptions are tested together (even though each may pass individually),
// some tests randomly fail. This is likely due to stochastic processes in initTestComponentsCommon,
// specifically in the setupRDKitFFWithPos call which ports RDKit's original ETKDG pipeline.
// Previous attempts to resolve similar issues by cleaning up the RDKit porting were partially
// successful but not definitive. Further investigation is needed
INSTANTIATE_TEST_SUITE_P(ETKDGOptions,
                         ETKDGMinimizeMultiMolDiverseTestFixture,
                         ::testing::Values(ETKDGOption::ETKDGv3),
                         [](const ::testing::TestParamInfo<ETKDGOption>& info) {
                           return getETKDGOptionName(info.param);
                         });
